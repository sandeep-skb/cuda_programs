
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <ctime>
using namespace std;

void host_init(float *arr, int n, float init_val){
	for (int i=0; i < n; i++)
		arr[i] = init_val;
}


void host_vec_add(float *h_a, float *h_b, float *h_c, int n){
	for (int i = 0; i < n; i++){
		h_c[i] = h_a[i] + h_b[i];
	}
}

__global__
void cuda_vec_add(float *d_a, float *d_b, float *d_c, int n, int m){
	int col = blockIdx.x*blockDim.x + threadIdx.x;
	int row = blockIdx.y*blockDim.y + threadIdx.y;
	if (col < m && row < n){
		d_c[row*m + col] = d_a[row*m + col] + d_b[row*m + col];
	}
}

void cudaError_check(hipError_t err){
	if (err != hipSuccess){
		printf("GPUassert: %s %s %d\n", hipGetErrorString(err), __FILE__, __LINE__);
		exit(EXIT_FAILURE);
	}
}

int main(){
	float *h_a, *h_b, *h_c, *h_c_cpy;
	float *d_a, *d_b, *d_c;
	int n = 1024; // row
	int m = 1024; // col

	size_t size = n*m*sizeof(float);
	clock_t start, stop;

//################## HOST Start ###################//
	h_a = (float*) malloc (size);
	h_b = (float*) malloc (size);
	h_c = (float*) malloc (size);
	h_c_cpy = (float*) malloc (size);


	host_init(h_a, n*m, 1);
	host_init(h_b, n*m, 2);
	host_init(h_c, n*m, 0);
	host_init(h_c_cpy, n*m, 0);
	start = clock();
	host_vec_add(h_a, h_b, h_c, n*m);
	stop = clock();
	double cpu_duration = (stop - start) / (double) CLOCKS_PER_SEC;
//################## HOST End ###################//

//################## CUDA Start ###################//
	hipError_t err ;
	err = hipMalloc((void **) &d_a, size);
	cudaError_check(err);
	err = hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	cudaError_check(err);
	err = hipMalloc((void **) &d_b, size);
	cudaError_check(err);
	err = hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
	cudaError_check(err);
	err = hipMalloc((void **) &d_c, size);
	cudaError_check(err);

	//Kernel invocation
	int num_threads_per_block = 256;
	dim3 gridDim  ((n-1)/num_threads_per_block + 1, (m-1)/num_threads_per_block + 1, 1);
	dim3 blockDim  (num_threads_per_block, num_threads_per_block, 1);
	start = clock();
	cuda_vec_add<<<gridDim, blockDim>>>(d_a, d_b, d_c, n, m);
	err = hipDeviceSynchronize();
	stop = clock();
	cudaError_check(err);
	err = hipMemcpy(h_c_cpy, d_c, size, hipMemcpyDeviceToHost);
	cudaError_check(err);
	double gpu_duration = (stop - start) / (double) CLOCKS_PER_SEC;
//################## CUDA End ###################//
	int success = 1;
	for (int i = 0; i < n*m; i++){
		if (h_c[i] != h_c_cpy[i]){
			success = 0;
			printf("Failure at idx: %d\n", i);
			break;
		}
	}
	if (success == 1)
		printf("Success\n");
	printf("CPU Duration: %0.3f secs \n", cpu_duration);
	printf("GPU Duration: %0.3f secs \n", gpu_duration);
	return 1;
}
