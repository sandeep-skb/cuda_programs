
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <ctime>
using namespace std;

#define TILE_WIDTH 16
void host_init(float *arr, int n, float init_val){
	for (int i=0; i < n; i++)
		arr[i] = init_val;
}


void host_matmul(float *h_a, float *h_b, float *h_c, int m, int n, int k){
	// h_a (mxk); h_b (kxn); h_c (mxn)
	for (int row = 0; row < m; row++){
		for (int col = 0; col < n; col++){
			float sum = 0;
			for (int kk = 0; kk < k; kk++){
				float a = h_a[row*k + kk];
				float b = h_b[kk*n + col];
				sum += a * b;
			}
			h_c[row*n + col] = sum;
		}
	}
}

__global__
void cuda_matmul(float *d_a, float *d_b, float *d_c, int m, int n, int k){
	__shared__ float shmem_a[TILE_WIDTH][TILE_WIDTH];
	__shared__ float shmem_b[TILE_WIDTH][TILE_WIDTH];
	int tx = threadIdx.x;
	int ty = threadIdx.y;
	int col = blockIdx.x*blockDim.x + tx;
	int row = blockIdx.y*blockDim.y + ty;
	float c_val = 0.0;
	for (int phase = 0; phase < (k-1)/TILE_WIDTH + 1; phase++){
		if (row < m && (tx + phase*TILE_WIDTH) < k)
			shmem_a[ty][tx] = d_a[row*k + (tx + phase*TILE_WIDTH)];
		else
			shmem_a[ty][tx] = 0.0;

		if (col < n && ((ty + phase*TILE_WIDTH) < k))
			shmem_b[ty][tx] = d_b[(ty + phase*TILE_WIDTH)*n + col];
		else
			shmem_b[ty][tx] = 0.0;


		__syncthreads();
		for (int kk = 0; kk < TILE_WIDTH; kk++){
			c_val += shmem_a[ty][kk] * shmem_b[kk][tx];
		}
		__syncthreads();
	}
	if (col < n and row < m)
		d_c[row*n + col] = c_val;
}

void cudaError_check(hipError_t err, int line){
	if (err != hipSuccess){
		printf("GPUassert: %s %s %d\n", hipGetErrorString(err), __FILE__, line);
		exit(EXIT_FAILURE);
	}
}

int main(){
	float *h_a, *h_b, *h_c, *h_c_cpy;
	float *d_a, *d_b, *d_c;
	// h_a dimensions = MxK
	// h_b dimensions = KxN
	// h_c dimensions = MxN
	int m = 1024; // 
	int n = 1024; // 
	int k = 1024; //

	size_t size_ha = k*m*sizeof(float);
	size_t size_hb = k*n*sizeof(float);
	size_t size_hc = m*n*sizeof(float);
	clock_t start, stop;

//################## HOST Start ###################//
	h_a = (float*) malloc (size_ha);
	h_b = (float*) malloc (size_hb);
	h_c = (float*) malloc (size_hc);
	h_c_cpy = (float*) malloc (size_hc);


	host_init(h_a, k*m, 1);
	host_init(h_b, n*k, 2);
	host_init(h_c, n*m, 0);
	host_init(h_c_cpy, n*m, 0);
	start = clock();
	host_matmul(h_a, h_b, h_c, m, n, k);
	stop = clock();
	double cpu_duration = (stop - start) / (double) CLOCKS_PER_SEC;
//################## HOST End ###################//

//################## CUDA Start ###################//
	hipError_t err ;
	err = hipMalloc((void **) &d_a, size_ha);
	cudaError_check(err, __LINE__);
	err = hipMemcpy(d_a, h_a, size_ha, hipMemcpyHostToDevice);
	cudaError_check(err, __LINE__);
	err = hipMalloc((void **) &d_b, size_hb);
	cudaError_check(err, __LINE__);
	err = hipMemcpy(d_b, h_b, size_hb, hipMemcpyHostToDevice);
	cudaError_check(err, __LINE__);
	err = hipMalloc((void **) &d_c, size_hc);
	cudaError_check(err, __LINE__);

	//Kernel invocation
	int num_threads_per_block = TILE_WIDTH;
	dim3 gridDim  ((m-1)/num_threads_per_block + 1, (n-1)/num_threads_per_block + 1, 1);
	dim3 blockDim  (num_threads_per_block, num_threads_per_block, 1);
	start = clock();
	cuda_matmul<<<gridDim, blockDim>>>(d_a, d_b, d_c, m, n, k);
	stop = clock();
	err = hipMemcpy(h_c_cpy, d_c, size_hc, hipMemcpyDeviceToHost);
	cudaError_check(err, __LINE__);
	double gpu_duration = (stop - start) / (double) CLOCKS_PER_SEC;
//################## CUDA End ###################//
	int success = 1;
	for (int i = 0; i < n*m; i++){
		if (h_c[i] != h_c_cpy[i]){
			success = 0;
			printf("Failure at idx: %d\n", i);
			break;
		}
	}
	if (success == 1)
		printf("Success\n");
	printf("CPU Duration: %0.3f secs \n", cpu_duration);
	printf("GPU Duration: %0.5f secs \n", gpu_duration);
	return 1;
}
